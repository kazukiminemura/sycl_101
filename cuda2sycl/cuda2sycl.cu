#include <hip/hip_runtime.h>
#include <iostream>

// ベクトル加算のCUDAカーネル
__global__ void vector_add(const float* A, const float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        std::cout << "No CUDA devices available." << std::endl;
        return 0;
    }

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        error = hipGetDeviceProperties(&deviceProp, device);
        std::cout << "Device " << device << ": " << deviceProp.name << std::endl;
    }

    int N = 1024;
    size_t bytes = N * sizeof(float);

    // ホストメモリの割り当て
    float* h_A = (float*)malloc(bytes);
    float* h_B = (float*)malloc(bytes);
    float* h_C = (float*)malloc(bytes);

    // データの初期化
    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // デバイスメモリの割り当て
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // データをデバイスに転送
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // カーネルの起動
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    vector_add<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    // 結果をホストにコピー
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    // 結果の検証
    for (int i = 0; i < N; i++) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            printf("エラー: インデックス %d\n", i);
            break;
        }
    }
    printf("計算完了\n");

    // メモリの解放
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
